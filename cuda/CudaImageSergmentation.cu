#include "hip/hip_runtime.h"
#include "CudaImageSergmentation.h"

hipDeviceProp_t prop;
int nMaxThreadsPerBlock = 1024;
char *dev_src3c, *dev_redMap, *dev_greenMap, *dev_yellowMap, *dev_ryMap;

__global__ void _segmentation(char *bgrIn, char *dev_redMap, char *dev_greenMap, char *dev_yellowMap,char *dev_ryMap, float *rgyRange, int width, int height)
{
	const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;	
	const int x = offset % width;
	const int y = (offset-x) / width;
		
	const int idx3c = y * ( width * 3 ) + x * 3;
	float bb = (unsigned char)bgrIn[ idx3c ];
	float gg = (unsigned char)bgrIn[ idx3c + 1];
	float rr = (unsigned char)bgrIn[ idx3c + 2];

	for( int i = 0; i < 3; i++ ) {		
		float diff = abs( rgyRange[(3*i)+0] - bb ) + abs( rgyRange[(3*i)+1] - gg ) + abs( rgyRange[(3*i)+2] - rr );		
		if( i == 0 ) {		// Red�� ��
			if( diff <= 80) { // 80
				dev_redMap[ offset ] = (unsigned char)255;
				dev_ryMap[ offset ] = (unsigned char)255;
			} else {
				dev_redMap[ offset ] = (unsigned char)0;
				dev_ryMap[ offset ] = (unsigned char)0;
			}
		} else if( i == 1 ) { // Cyan�� ��, 90
			if( diff <= 180.0f ) dev_greenMap[ offset ] = (unsigned char)255;
			else dev_greenMap[ offset ] = (unsigned char)0;
		
		} else if( i == 2 ) {
			if( diff <= 85 ) { // Yellow�� ��, 80
				dev_yellowMap[ offset ] = (unsigned char)255;				
				dev_ryMap[ offset ] = (unsigned char)255;
			} else {
				dev_yellowMap[ offset ] = (unsigned char)0;
				//dev_ryMap[ offset ] = (unsigned char)0;
			}
		}
	}
	
}

__global__ void _hsvSegmentation(char *bgrIn, char *dev_redMap, char *dev_greenMap, char *dev_yellowMap,char *dev_ryMap, float *rgyRange, int width, int height)
{
	const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;	
	const int x = offset % width;
	const int y = (offset-x) / width;
		
	const int idx3c = y * ( width * 3 ) + x * 3;
	float hue = (unsigned char)bgrIn[ idx3c ];
	float sat = (unsigned char)bgrIn[ idx3c + 1];
	float bright = (unsigned char)bgrIn[ idx3c + 2];

	if( bright < 200 ) {
		dev_redMap[ offset ] = (unsigned char)0;
		dev_ryMap[ offset ] = (unsigned char)0;
		dev_greenMap[ offset ] = (unsigned char)0;
		return;
	} else if( bright >= 250 ) {
		dev_redMap[ offset ] = (unsigned char)0;
		dev_ryMap[ offset ] = (unsigned char)0;
		dev_greenMap[ offset ] = (unsigned char)0;
		return;
	}

	for( int i = 0; i < 4; i++ ) {		
		
		float diff = abs( rgyRange[(3*i)+0] - hue ) + abs( rgyRange[(3*i)+1] - sat );
		float hueDiff = abs( rgyRange[(3*i)+0] - hue );
		float satDiff = abs( rgyRange[(3*i)+1] - sat );
		if( i == 0 ) {									
			if( hueDiff <= 40 ) {
				dev_redMap[ offset ] = (unsigned char)255;
				dev_ryMap[ offset ] = (unsigned char)255;
			} else {
				dev_redMap[ offset ] = (unsigned char)0;
				dev_ryMap[ offset ] = (unsigned char)0;
			}
		} else if( i == 2 ) {
			if( diff <= 40 ) dev_greenMap[ offset ] = (unsigned char)255;
			else dev_greenMap[ offset ] = (unsigned char)0;
		
		} else if( i == 3 ) {
			if( hueDiff <= 20 ) {
				dev_yellowMap[ offset ] = (unsigned char)255;				
				dev_ryMap[ offset ] = (unsigned char)255;
			} else {
				dev_yellowMap[ offset ] = (unsigned char)0;
				dev_ryMap[ offset ] = (unsigned char)0;
			}
		}
	}
	
}

void initSegmentationModule(int width, int height)
{	
	hipDeviceProp_t prop;
	int nMaxThreadsPerBlock;

	int count = 1;
	hipGetDeviceCount( &count );
	for( int i = 0; i < count; i++  ) {
		hipGetDeviceProperties( &prop, i );
		nMaxThreadsPerBlock = prop.maxThreadsPerBlock;		
	}

	hipMalloc( (void**)&dev_src3c, sizeof(char)*width*height*3 );
	hipMalloc( (void**)&dev_redMap, sizeof(char)*width*height );
	hipMalloc( (void**)&dev_greenMap, sizeof(char)*width*height );
	hipMalloc( (void**)&dev_yellowMap, sizeof(char)*width*height );
	hipMalloc( (void**)&dev_ryMap, sizeof(char)*width*height );
}

void freeSegmentationModule()
{	
	hipFree( dev_src3c );
	hipFree( dev_redMap );
	hipFree( dev_greenMap );
	hipFree( dev_yellowMap );
}

void gpu_segmentation( char *host_src3c, char *host_redMap, char *host_greenMap, char *host_yellowMap, char *host_ryMap, float* rgyRange, int width, int height )
{	
	const int nBlocks = ( width * height + nMaxThreadsPerBlock - 1 )  / nMaxThreadsPerBlock;	
	
	float *dev_rgyRange;
	hipMalloc( (void**)&dev_rgyRange, sizeof(float)*9 );
	hipMemcpy( dev_src3c, host_src3c, sizeof(char)*width*height*3, hipMemcpyHostToDevice );
	hipMemcpy( dev_rgyRange, rgyRange, sizeof(float)*9, hipMemcpyHostToDevice );

	_segmentation<<< nBlocks, nMaxThreadsPerBlock >>>( dev_src3c, dev_redMap, dev_greenMap, dev_yellowMap, dev_ryMap, dev_rgyRange, width, height );

	hipMemcpy( host_redMap, dev_redMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	hipMemcpy( host_greenMap, dev_greenMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	hipMemcpy( host_yellowMap, dev_yellowMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	hipMemcpy( host_ryMap, dev_ryMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	
	hipFree( dev_rgyRange );

}

void gpu_hsvSegmentation( char *host_src3c, char *host_redMap, char *host_greenMap, char *host_yellowMap, char *host_ryMap, float* rgyRange, int width, int height )
{	
	const int nBlocks = ( width * height + nMaxThreadsPerBlock - 1 )  / nMaxThreadsPerBlock;	
	
	float *dev_rgyRange;
	hipMalloc( (void**)&dev_rgyRange, sizeof(float)*12 );
	hipMemcpy( dev_src3c, host_src3c, sizeof(char)*width*height*3, hipMemcpyHostToDevice );
	hipMemcpy( dev_rgyRange, rgyRange, sizeof(float)*9, hipMemcpyHostToDevice );

	_hsvSegmentation<<< nBlocks, nMaxThreadsPerBlock >>>( dev_src3c, dev_redMap, dev_greenMap, dev_yellowMap, dev_ryMap, dev_rgyRange, width, height );

	hipMemcpy( host_redMap, dev_redMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	hipMemcpy( host_greenMap, dev_greenMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	hipMemcpy( host_yellowMap, dev_yellowMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	hipMemcpy( host_ryMap, dev_ryMap, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	
	hipFree( dev_rgyRange );

}
