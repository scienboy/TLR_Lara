#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void _erode(char *grayIn, char *grayOut, int width, int height, int kernelSize, int *mask)
{
	const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;	
	const int x = offset % width;
	const int y = (offset-x) / width;

	if( x - kernelSize / 2 < 0 || y - kernelSize / 2 < 0 || x + kernelSize / 2 > width || y + kernelSize / 2 > height ) return;

	int intensitySum = 0;	
	int subImageSize = kernelSize * kernelSize;
	
	for( int i = 0; i < subImageSize; i++ ) {		
		if( (unsigned char)grayIn[ offset + mask[i] ] > 100 ) intensitySum++;
	}
	
	if( intensitySum == subImageSize )
		grayOut[ offset ] = (unsigned char)255;
	else
		grayOut[ offset ] = (unsigned char)0;

}

__global__ void _dilate(char *grayIn, char *grayOut, int width, int height, int kernelSize, int *mask)
{
	const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;	
	const int x = offset % width;
	const int y = (offset-x) / width;

	if( x - kernelSize / 2 < 0 || y - kernelSize / 2 < 0 || x + kernelSize / 2 > width || y + kernelSize / 2 > height ) return;

	int intensitySum = 0;	
	int subImageSize = kernelSize * kernelSize;
	
	for( int i = 0; i < subImageSize; i++ ) {		
		if( (unsigned char)grayIn[ offset + mask[i] ] > 100 ) intensitySum++;
	}
	
	if( intensitySum >= 1 )
		grayOut[ offset ] = (unsigned char)255;
	else
		grayOut[ offset ] = (unsigned char)0;
	
}

int* _getRangeMask(int kernelSize, int width)
{	
	int *mask = (int*)malloc(sizeof(int)*kernelSize*kernelSize);
	int subImageSize = kernelSize*kernelSize;
	int centerPixel = subImageSize / 2;
	int centerPixelCol = centerPixel / ( subImageSize / kernelSize );
	int centerPixelRow = centerPixel % ( subImageSize / kernelSize );	
	for( int i = 0; i < subImageSize; i++ ) {
		int rowIdx = i / ( subImageSize / kernelSize );
		int colIdx = i % ( subImageSize / kernelSize );		
		mask[i] = ( rowIdx - centerPixelRow ) * width + ( colIdx - centerPixelCol );
	}

	return mask;
}

void gpu_erode( char *host_src, char *host_dst, int width, int height, int kernelSize = 3 )
{
	hipDeviceProp_t prop;
	int nMaxThreadsPerBlock;
	int count = 1;
	hipGetDeviceCount( &count );
	for( int i = 0; i < count; i++  ) {
		hipGetDeviceProperties( &prop, i );
		nMaxThreadsPerBlock = prop.maxThreadsPerBlock;		
	}
	const int nBlocks = ( width * height + nMaxThreadsPerBlock - 1 )  / nMaxThreadsPerBlock;	
	char *dev_src, *dev_dst;
	hipMalloc( (void**)&dev_src, sizeof(char)*width*height );
	hipMalloc( (void**)&dev_dst, sizeof(char)*width*height );
	hipMemcpy( dev_src, host_src, sizeof(char)*width*height, hipMemcpyHostToDevice );

	int *host_mask = _getRangeMask( kernelSize, width );
	int *dev_mask;
	hipMalloc( (void**)&dev_mask, sizeof(int)*kernelSize*kernelSize);
	hipMemcpy( dev_mask, host_mask, sizeof(int)*kernelSize*kernelSize, hipMemcpyHostToDevice );

	_erode<<< nBlocks, nMaxThreadsPerBlock >>>( dev_src, dev_dst, width, height, kernelSize, dev_mask );
	
	hipMemcpy( host_dst, dev_dst, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	
	hipFree( dev_src );
	hipFree( dev_dst );
	hipFree( dev_mask );
	free( host_mask );
}

void gpu_dilate( char *host_src, char *host_dst, int width, int height, int kernelSize = 3 )
{
	hipDeviceProp_t prop;
	int nMaxThreadsPerBlock;
	int count = 1;
	hipGetDeviceCount( &count );
	for( int i = 0; i < count; i++  ) {
		hipGetDeviceProperties( &prop, i );
		nMaxThreadsPerBlock = prop.maxThreadsPerBlock;		
	}
	const int nBlocks = ( width * height + nMaxThreadsPerBlock - 1 )  / nMaxThreadsPerBlock;	
	char *dev_src, *dev_dst;
	hipMalloc( (void**)&dev_src, sizeof(char)*width*height );
	hipMalloc( (void**)&dev_dst, sizeof(char)*width*height );
	hipMemcpy( dev_src, host_src, sizeof(char)*width*height, hipMemcpyHostToDevice );
	
	int *host_mask = _getRangeMask( kernelSize, width );
	int *dev_mask;
	hipMalloc( (void**)&dev_mask, sizeof(int)*kernelSize*kernelSize);
	hipMemcpy( dev_mask, host_mask, sizeof(int)*kernelSize*kernelSize, hipMemcpyHostToDevice );
		
	_dilate<<< nBlocks, nMaxThreadsPerBlock >>>( dev_src, dev_dst, width, height, kernelSize, dev_mask );
	
	hipMemcpy( host_dst, dev_dst, sizeof(char)*width*height, hipMemcpyDeviceToHost );
	
	
	hipFree( dev_src );
	hipFree( dev_dst );
	hipFree( dev_mask );
	free( host_mask );
}